
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <chrono>


__global__
void multiplyCell(int N, int * a, int * b, int * c){

    // We get the index of the current data 
    unsigned int threadx = blockDim.x * blockIdx.x + threadIdx.x;
    unsigned int thready = threadIdx.y + blockIdx.y * blockDim.y;
    unsigned int threadxy = thready * N + threadx;

    // Then we get the col and row
    int row = threadxy / N;
    int col = threadxy % N;

    if(row < N && col < N){

        // Then we multiply and add each one of them
        int result = 0;
        for(int i=0;i<N;i++){
            result +=a[row*N+i]*b[i*N+col];
        }

        c[threadx]=result;

    }

}

void GPUTimedMatrixMultiplication(int N,int * a,int * b, int * c,
        int ** runs, int runsLength){

    // Allocate in GPU
    int *d_a,*d_b,*d_c;
    int size = N*N*sizeof(int);
    hipMalloc(&d_a,size);
    hipMalloc(&d_b,size);
    hipMalloc(&d_c,size);

    // Transfer to device
    hipMemcpy(d_a,a,size,hipMemcpyHostToDevice);
    hipMemcpy(d_b,b,size,hipMemcpyHostToDevice);

    // Call kernel with the blocks, grid and threads specified
    for(int i=0;i<runsLength;i++){

        int * run = runs[i];
        dim3 blocksPerGrid(run[0],run[1],run[2]);
        dim3 threadsPerBlock(run[3],run[4],run[5]);

        //initialize timer
        auto start = std::chrono::high_resolution_clock::now();
        multiplyCell<<<blocksPerGrid,threadsPerBlock>>>(N,d_a,d_b,d_c); 
        //finish timer
        auto end = std::chrono::high_resolution_clock::now();

        std::chrono::duration<float, std::milli> duration_ms = end - start;

        //print result
        printf("GPU test dimensions threads %d %d blocks %d %d N: %d duration: %f\n ms\n",
                run[0],run[1],run[3],run[4],N,duration_ms.count());
        fflush(stdout); 
    }

    // Copy result back from gpu
    hipMemcpy(c,d_c,size,hipMemcpyDeviceToHost);

    // Free variables
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

}

void GPUMatrixMultiplication(int N,int * a,int * b, int * c,
        int * run){

    // Allocate in GPU
    int *d_a,*d_b,*d_c;
    int size = N*N*sizeof(int);
    hipMalloc(&d_a,size);
    hipMalloc(&d_b,size);
    hipMalloc(&d_c,size);

    // Transfer to device
    hipMemcpy(d_a,a,size,hipMemcpyHostToDevice);
    hipMemcpy(d_b,b,size,hipMemcpyHostToDevice);

    // Call kernel with the blocks, grid and threads specified
    dim3 blocksPerGrid(run[0],run[1],run[2]);
    dim3 threadsPerBlock(run[3],run[4],run[5]);
    multiplyCell<<<blocksPerGrid,threadsPerBlock>>>(N,d_a,d_b,d_c); 

    // Copy result back from gpu
    hipMemcpy(c,d_c,size,hipMemcpyDeviceToHost);

    // Free variables
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

}
